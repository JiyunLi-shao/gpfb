#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <err.h>

const double PI = 3.14159265358979323846;

const double FS = 1024;//MHz
const unsigned FRAMES = 50;

float sinc(float x)
{
    if (x == 0.0f)
        return 1.0;
    else
        return sin(x)/x;
}

float *gen_fir(float *buf, unsigned taps, float fc)
{
    assert(!(taps%2));

    for(size_t i=0; i<taps; ++i)
        buf[i] = sinc(PI*fc*(i-taps/2.0))*fc;
    return buf;
}


//Generate random noise normalized to (-norm..norm)/2
float *gen_rand(float *buf, size_t N, float norm)
{
    for(size_t i=0;i<N;++i)
        buf[i] += rand()*norm/RAND_MAX - norm/2.0;
    return buf;
}

//Generate impulse
float *gen_imp(float *buf, size_t N)
{
    (void) N;
    *buf += 1.0;
    return buf;
}

//Generate sawtooth wave with given period in samples
float *gen_saw(float *buf, size_t N, size_t period)
{
    const float low = period/-2.0;
    float state     = low;
    for(size_t i=0;i<N;++i)
        buf[i] += (state = i%period ? state+1.0 : low)/-low;
    return buf;
}

//Generate dc offset
float *gen_dc(float *buf, size_t N)
{
    for(size_t i=0;i<N;++i)
        buf[i] += 1.0;
    return buf;
}

//generate sin wave at frequency fq
float *gen_sin(float *buf, size_t N, float fq)
{
    const float rate = 2.0*PI*fq/FS;
    for(size_t i=0;i<N;++i) //TODO change sin to cos after testing
        buf[i] += cos(rate*i);
    return buf;
}

//TODO update function to new conventions
float *gen_chirp(float *buf, size_t N, size_t period, double dr)
{
    double rate = 2.0*PI/period,
           state = 0;
    for(size_t i=0;i<N;++i,state+=rate,rate+=dr)
        buf[i] += sin(state);
    return buf;
}

__global__ void convolve(float *coeff, size_t N, float *src, size_t M, float *dest, size_t chans)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i>M) 
        return;

    //do actual work at i
    dest[i] = 0.0;
    for(size_t j=0; j<N; ++j)
        if(j%chans==0)
            dest[i] += src[i-j]*coeff[j];
}

#undef checked
#define checked(x) { if(x!=hipSuccess) err(1, #x);}
float *apply_fir(float *buf, size_t N, float *coeff, size_t M, size_t chans)
{
    //insure clean decimation
    assert(M%chans==0);
    assert(N%chans==0);

    float *cu_coeff=NULL, *cu_buf=NULL, *cu_smps=NULL;//r,r,w

    //Allocate
    puts("Allocating...");
    checked(hipMalloc((void **)&cu_coeff, M*sizeof(float)));
    checked(hipMalloc((void **)&cu_smps, N*sizeof(float)));
    checked(hipMalloc((void **)&cu_buf, (N+M)*sizeof(float)));

    //Send
    puts("Sending...");
    checked(hipMemcpy(cu_coeff, coeff, M*sizeof(float), hipMemcpyHostToDevice));
    checked(hipMemcpy(cu_buf, buf-M, (N+M)*sizeof(float), hipMemcpyHostToDevice));

    //Run
    puts("Running...");
    int block_size = 128;
    int blocks = N/block_size + (N%block_size == 0 ? 0:1);
    convolve <<< blocks, block_size >>>(cu_coeff, M, cu_buf+M, N, cu_smps, chans);

    //Retreive
    puts("Getting...");
    checked(hipMemcpy(buf, cu_smps, sizeof(float)*N, hipMemcpyDeviceToHost));

    //Clean
    puts("Cleaning...");
    checked(hipFree(cu_coeff));
    checked(hipFree(cu_smps));
    checked(hipFree(cu_buf));
    return buf;

}

int main()
{
    const size_t CHANNELS = 4,
                 N        = CHANNELS*8;
    float fir[N];
    gen_fir(fir, N, 1.0/CHANNELS);

    const size_t M=N*128;
    float buf[M+N];
    memset(buf, 0, N*sizeof(float));
    float *smps=buf+N;
    //gen_rand(smps, M, 1.0);
    gen_saw(smps, M, 1024);

    //Show previous
    FILE *fb = fopen("before.txt", "w+");
    for(size_t i=0;i<M;++i)
        fprintf(fb, "%f, ", smps[i]);
    fclose(fb);

    //Apply to samples
    apply_fir(smps, M, fir, N, CHANNELS);

    //Show results
    FILE *fa = fopen("after.txt", "w+");
    for(size_t i=0;i<M;++i)
        fprintf(fa, "%f, ", smps[i]);
    fclose(fa);
    return 0;
}
