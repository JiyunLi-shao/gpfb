#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <assert.h>
#include <err.h>

const double PI = 3.14159265358979323846;

const double FS = 1024;//MHz
const unsigned FRAMES = 50;

float sinc(float x)
{
    if (x == 0.0f)
        return 1.0;
    else
        return sin(x)/x;
}

/* Put fir coeffs into buf with length taps
 * The filter has a corner frequency at fc (normalized)
 */
float *gen_fir(float *buf, unsigned taps, float fc)
{
    assert(!(taps%2));

    for(size_t i=0; i<taps; ++i)
        buf[i] = sinc(PI*fc*(i-taps/2.0))*fc;
    return buf;
}

//Rescale FIR filter coeff
float *scale_fir(float *buf, unsigned N)
{
    float max = 0.0;
    for(size_t i=0; i<N; ++i)
        max = buf[i]>max?buf[i]:max;

    for(size_t i=0; i<N; ++i)
        buf[i] /= max;
    return buf;
}


//Generate random noise normalized to (-norm..norm)/2
float *gen_rand(float *buf, size_t N, float norm)
{
    for(size_t i=0;i<N;++i)
        buf[i] += rand()*norm/RAND_MAX - norm/2.0;
    return buf;
}

//Generate impulse
float *gen_imp(float *buf, size_t N)
{
    (void) N;
    *buf += 1.0;
    return buf;
}

//Generate sawtooth wave with given period in samples
float *gen_saw(float *buf, size_t N, size_t period)
{
    const float low = period/-2.0;
    float state     = low;
    for(size_t i=0;i<N;++i)
        buf[i] += (state = i%period ? state+1.0 : low)/-low;
    return buf;
}

//Generate dc offset
float *gen_dc(float *buf, size_t N)
{
    for(size_t i=0;i<N;++i)
        buf[i] += 1.0;
    return buf;
}

//generate sin wave at frequency fq
float *gen_sin(float *buf, size_t N, float fq)
{
    const float rate = 2.0*PI*fq/FS;
    for(size_t i=0;i<N;++i) //TODO change sin to cos after testing
        buf[i] += cos(rate*i);
    return buf;
}

//TODO update function to new conventions
float *gen_chirp(float *buf, size_t N, size_t period, double dr)
{
    double rate = 2.0*PI/period,
           state = 0;
    for(size_t i=0;i<N;++i,state+=rate,rate+=dr)
        buf[i] += sin(state);
    return buf;
}

//Complex to real
float to_real(float x, float y)
{
    return sqrt(x*x+y*y);
}

#define checked(x) { hipfftResult_t e = x; if(e) err(e, #x);}
//apply fft using out of place transform
float *apply_fft(float *src, float *dest, size_t transform_size, size_t batches)
{
    printf("Samples to fft: %ld\n", batches*transform_size);
    assert(HIPFFT_SUCCESS==0);
    // Setup
    hipfftHandle plan;
    checked(hipfftPlan1d(&plan, transform_size, HIPFFT_R2C, batches));

    // Perform FFT
    checked(hipfftExecR2C(plan, src, (hipfftComplex *)dest));

    // Cleanup
    checked(hipfftDestroy(plan));

    return dest;
}

__global__ void convolve(float *coeff, size_t N, float *src, size_t M, float *dest, size_t chans)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i>M)
        return;

    unsigned sel = i%chans;

    //do actual work at i
    dest[i] = 0.0;
    for(size_t j=sel; j<N; j+=chans)
        dest[i] += src[i-sel-j]*coeff[j];
}

#undef checked
#define checked(x) { if(x!=hipSuccess) err(1, #x);}
float *apply_fir(float *buf, size_t N, float *coeff, size_t M, size_t chans)
{
    //insure clean decimation
    assert(M%chans==0);
    assert(N%chans==0);

    float *cu_coeff=NULL, *cu_buf=NULL, *cu_smps=NULL;//r,r,w

    //Allocate
    puts("Allocating...");
    checked(hipMalloc((void **)&cu_coeff, M*sizeof(float)));
    checked(hipMalloc((void **)&cu_smps, N*sizeof(float)));
    checked(hipMalloc((void **)&cu_buf, (N+M)*sizeof(hipfftComplex)));

    //Send
    puts("Sending...");
    checked(hipMemcpy(cu_coeff, coeff, M*sizeof(float), hipMemcpyHostToDevice));
    checked(hipMemcpy(cu_buf, buf-M, (N+M)*sizeof(float), hipMemcpyHostToDevice));

    //Run
    puts("Filtering...");
    int block_size = 128;
    int blocks = N/block_size + (N%block_size == 0 ? 0:1);
    convolve <<< blocks, block_size >>>(cu_coeff, M, cu_buf+M, N, cu_smps, chans);
    hipDeviceSynchronize();

    //Post Process
    puts("FFT...");
    apply_fft(cu_smps, cu_buf, chans, N/chans);
    hipDeviceSynchronize();

    //Retreive
    puts("Getting...");
    checked(hipMemcpy(buf, cu_buf, sizeof(float)*N, hipMemcpyDeviceToHost));

    //Clean
    puts("Cleaning...");
    checked(hipFree(cu_coeff));
    checked(hipFree(cu_smps));
    checked(hipFree(cu_buf));
    return buf;

}

int main()
{
    const size_t CHANNELS = 8,
                 N        = CHANNELS*16;
    float fir[N];
    gen_fir(fir, N, 0.5/CHANNELS);
    scale_fir(fir, N);

    const size_t M=N*FRAMES;
    float buf[M+N];
    memset(buf, 0, (M+N)*sizeof(float));
    float *smps=buf+N;
    //gen_rand(smps, M, 1.0);
    //gen_saw(smps, M, 278);
    gen_chirp(smps, M, 1024*16, 0.0008);
    //gen_sin(smps, M, 8);
    //gen_sin(smps, M, 16);
    gen_sin(smps, FS, 500.0);
    //gen_imp(smps,M);

    //Show previous
    FILE *fb = fopen("before.txt", "w+");
    for(size_t i=0;i<M;++i)
        fprintf(fb, "%f, ", smps[i]);
    fclose(fb);

    //Apply to samples
    apply_fir(smps, M, fir, N, CHANNELS);

    //Show results
    FILE *fa = fopen("after.txt", "w+");
    float2 *out = (float2*)smps;
    for(size_t i=0;i<M/2;i++) {
        float smp = to_real(out[i].x, out[i].y);
        fprintf(fa, "%c%f", i?',':' ', smp);
    }
    fclose(fa);
    return 0;
}
