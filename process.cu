#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <assert.h>
#include <stdint.h>
#include <math.h>
#include "siggen.h"
#include "process.h"
#include <err.h>
#include <iostream>

//Function Declarations
void apply_polyphase(int8_t *buf, Pfb &pfb, float *hack=NULL);

//Macro Definitions
#define checked(x) { hipError_t e = x; if(e!= hipSuccess) {\
    fprintf(stderr,"CUDA error[%s][%d]: %s\n", #x, __LINE__, hipGetErrorString(e));\
    exit(-1);}}

#define check_launch { hipError_t e = hipGetLastError(); if(e!= hipSuccess) {\
    fprintf(stderr,"CUDA launch error[%d]: %s\n", __LINE__, hipGetErrorString(e));\
    exit(-1);}}

#define fftchecked(x) { hipfftResult_t e = x; if(e) {\
    fprintf(stderr,"CUFFT error[%s][%d]: %s\n", #x, e,\
            hipGetErrorString(hipGetLastError()));\
    exit(0);}}

//Class Definitions
struct Pfb
{
    void i_pfb(const float *fir, size_t _nFir, size_t _nSmps, size_t _nChan);
    void d_pfb(void);

    //Call when device buffer is unused
    void run(int8_t *data, float *hack=NULL) {
        apply_polyphase(data, *this, hack);
    };

    void sync(void) {
        checked(hipStreamSynchronize(stream));
    }

    //Parameter sizes
    size_t nFir, nSmps, nChan;

    //Device Buffers
    int8_t *bitty;
    float   *fir,
            *buf,
            *smps;

    //Host buffer
    int8_t *h_buf;

    //FFT handle
    hipfftHandle plan;

    //stream handle for execution flow
    hipStream_t stream;
};

void Pfb::i_pfb(const float *_fir, size_t _nFir, size_t _nSmps, size_t _nChan)
{
    nFir  = _nFir;
    nSmps = _nSmps;
    nChan = _nChan;

    //insure clean decimation
    assert((nFir  % nChan) == 0);
    assert((nSmps % nChan) == 0);

    //Allocate GPU buffers
    checked(hipMalloc((void **)&bitty, nSmps));
    checked(hipMalloc((void **)&fir, nFir*sizeof(float)));
    size_t nStrideData = nSmps+2*nSmps/nChan;
    checked(hipMalloc((void **)&smps,(nStrideData+nFir)*sizeof(float)));
    checked(hipMalloc((void **)&buf, (nStrideData+nFir)*sizeof(float)));

    //Send over FIR data [TODO re-evaluate for const memory]
    checked(hipStreamCreate(&stream));
    checked(hipMemcpyAsync(fir, _fir, nFir*sizeof(float),
                hipMemcpyHostToDevice, stream));

    //Allocate CPU buffer
    checked(hipHostAlloc((void**) &h_buf, nSmps, hipHostMallocDefault));

    //Allocate FFT
#define CUFFT_LIMIT (1<<27)
    if(nSmps > CUFFT_LIMIT)
        fprintf(stderr, "Warning: CUFFT_LIMIT exceeded, please reduce "
                "batches\n");
    // Setup
    fftchecked(hipfftPlan1d(&plan, nChan, HIPFFT_R2C, nSmps/nChan));
    //fftchecked(cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE));

    fftchecked(hipfftSetStream(plan, stream));
}

void Pfb::d_pfb(void)
{
    //Clean up gpu memory
    checked(hipFree(bitty));
    checked(hipFree(fir));
    checked(hipFree(smps));
    checked(hipFree(buf));

    //Clean up host memory
    checked(hipHostFree(h_buf));

    // Cleanup
    fftchecked(hipfftDestroy(plan));

    hipStreamDestroy(stream);
}

std::ostream &operator<<(std::ostream &out, const Pfb &p)
{
    using namespace std;
#define prnt(x) out << #x ": " << (void *) x << endl;
    prnt(p.nFir);
    prnt(p.nSmps);
    prnt(p.nChan);
    prnt(p.bitty);
    prnt(p.fir);
    prnt(p.buf);
    prnt(p.smps);
    return out;
}

class Pfb *alloc_pfb(const float *fir, size_t _nFir, size_t _nSmps, size_t _nChan)
{
    Pfb *p = new Pfb;
    p->i_pfb(fir, _nFir, _nSmps, _nChan);
    return p;
}

void delete_pfb(class Pfb *p)
{
    p->d_pfb();
    delete p;
}

//Helper Functions
void show_mem_header(void)
{
    puts("free\ttotal\tused");
}
void show_mem(void)
{
    float free_m,total_m,used_m;
    size_t free,total;

    hipMemGetInfo((size_t*)&free,(size_t*)&total);

    free_m =free/1048576.0 ;
    total_m=total/1048576.0;
    used_m=(total_m-free_m);
    printf ( "%f\t%f\t%f\n", free_m,total_m,used_m);
}

//assumes smps -> buf
void fft_pad(Pfb &pfb)
{
    //Use Aliases for clarity
    const size_t width  = pfb.nChan,
          height = pfb.nSmps/pfb.nChan,
          sFloat = sizeof(float);

    float *dest = pfb.buf;
    const float *src = pfb.smps;

    checked(hipMemcpy2DAsync(dest, (width+2)*sFloat, src, width*sFloat,
                width*sFloat, height, hipMemcpyDeviceToDevice, pfb.stream));
}

//apply fft using out of place transform
float *apply_fft(float *src, float *dest, Pfb &pfb)
{
    fft_pad(pfb);
    // Perform FFT
    fftchecked(hipfftExecR2C(pfb.plan, src, (hipfftComplex *)dest));
    return dest;
}

//Main Kernel code

//Location is base address + offset
#define LOC const size_t i =\
                            (gridDim.y*(blockIdx.y*gridDim.x+blockIdx.x))+threadIdx.x

__global__ void cu_quantize(int8_t *dest, const float *src, size_t N, size_t
        chans)
{
    LOC;
    if(i<N)
        dest[i] = (static_cast<int8_t>(src[i]*7.5f)&0xc0);//0x00);//src[i]/32);//*128.0);
}

__global__ void cu_unquantize(float *dest, const int8_t *src, size_t N)
{
    LOC;
    if(i<N)
        dest[i] = src[i];
}

//half compression function
__device__ inline int8_t hcomp(int32_t d)
{
    return (d&0xc0000000>>24)|(d&0xc00000>>18)|(d&0xc000>>12)|(d&0xc0>>6);
}

__global__ void cu_compress(int8_t *data)
{
    LOC;
    //warning thread sync issues?
    int32_t chunk = ((int32_t*) data)[i];
    data[i] = hcomp(chunk);
}

__global__ void cu_stripper(int8_t *dest, const int8_t *src, size_t destWidth, size_t bytes)
{
    LOC;
    size_t srcWidth = destWidth+4;
    //transform source
    size_t j = i%destWidth + (i/destWidth)*srcWidth;
    //if(j < bytes)
    size_t N = destWidth+4;
#if 0
    if((i%N && (i+1)%N))
        dest[i] = src[i-2];
#else
    dest[i] = src[j];//&0xc0;
#endif
}

__global__ void convolve(float *dest, const float *src, const float *coeff,
        size_t nC, size_t nS, size_t chan)
{
    LOC;
    if (i<nS) {
        unsigned     sel   = i%chan;

        //do actual work at i
        float result = 0.0f;
#pragma unroll
        for(size_t j=sel; j<nC; j+=chan)
            result += src[i-sel-j]*coeff[j];
        dest[i] = result/chan;
    }
}

void apply_polyphase(int8_t *buf, Pfb &pfb, float *hack)
{
    checked(hipMemcpyAsync(pfb.bitty, buf, pfb.nSmps, hipMemcpyHostToDevice, pfb.stream));
    //Buffer with zeros
    checked(hipMemsetAsync(pfb.buf, 0, pfb.nFir*sizeof(float), pfb.stream));

/* FIXME these could be changed with device/cuda version
 *       get dynamically from deviceinfo API
 *       blocks: 1024x1024x64
 *       grids:  65535x65535x65535
 */
#define MAX_BLOCK 1<<10
//using powers of 2
#define MAX_GRID 1<<15
//macro assumes variables/literals as input
#define div_up(top,bot) ((top)/(bot)+((top)%(bot)==0 ? 0:1))

    //Run
    //puts("Filtering...");
    //Calculate dimensions
    const size_t block_x = MAX_BLOCK,
                 grid_y = pfb.nSmps/MAX_BLOCK > MAX_BLOCK ? MAX_BLOCK : 1,
                 grid_x  = div_up(pfb.nSmps,block_x*grid_y);
    const dim3   block(block_x, 1, 1),
                 grid(grid_x, grid_y, 1);

    if(0)
        printf("thread(%d, %d, %d) block(%d, %d, %d) for %ld elms.\n",
                block.x, block.y, block.z,
                grid.x,  grid.y,  grid.z, pfb.nSmps);

    //Convert to floating point
    cu_unquantize<<<grid, block, 0, pfb.stream>>>(pfb.buf+pfb.nFir, pfb.bitty, pfb.nSmps);
    check_launch;

    convolve<<<grid, block, 0, pfb.stream>>>(pfb.smps, pfb.buf+pfb.nFir, pfb.fir, pfb.nFir,
            pfb.nSmps, pfb.nChan);
    check_launch;

    //Post Process
    apply_fft(pfb.buf, pfb.buf, pfb);
    check_launch;

    //Provide all channels with full precision
    if(hack) checked(hipMemcpyAsync(hack, pfb.buf, pfb.nSmps*sizeof(float), hipMemcpyDeviceToHost, pfb.stream));

    //Convert to fixed point
    cu_quantize<<<grid, block, 0, pfb.stream>>>(pfb.bitty, pfb.buf, pfb.nSmps, pfb.nChan);
    check_launch;

    //Retreive
#if 0
    //checked(hipMemcpyAsync(buf, pfb.bitty, pfb.nSmps, hipMemcpyDeviceToHost, pfb.stream));
#else
    size_t width = pfb.nChan - 2;
    //printf("destination: %p\n", buf);
    //printf("width:       %lu\n", width);
    //printf("source:      %p\n", pfb.bitty+1);
    //printf("src stride:  %lu\n", width+2);
    //printf("height:      %lu\n", pfb.nSmps/width);

    checked(hipMemsetAsync(pfb.smps, 0, pfb.nSmps, pfb.stream));

    //no not that kind
    cu_stripper<<<grid, block, 0, pfb.stream>>>((int8_t *)pfb.smps,
            pfb.bitty+2, width, pfb.nSmps);
    //checked(hipMemcpy2DAsync(pfb.smps, width, pfb.bitty+1, width+4, width,
    //            pfb.nSmps/(width+4), hipMemcpyDeviceToDevice, pfb.stream));
    //TODO get right length
    //checked(hipMemcpyAsync(pfb.smps, pfb.bitty, pfb.nSmps, hipMemcpyDeviceToDevice, pfb.stream));
    checked(hipMemcpyAsync(buf, pfb.smps, pfb.nSmps+0*width*pfb.nSmps/(width+4), hipMemcpyDeviceToHost, pfb.stream));
#endif

#if 0
    dim3 nblock = block;
    nblock.x /= 4;
    cu_compress<<<grid, nblock, 0, pfb.stream>>>(pfb.bitty);

#endif
}

void apply_pfb_direct(int8_t *buffer, Pfb *p)
{
    p->run((int8_t*)buffer);
}

void sync_pfb_direct(Pfb *p)
{
    p->sync();
}

void apply_pfb(float *buffer, Pfb *p)
{
    const size_t N = p->nSmps;
    int8_t *buf   = p->h_buf;
    apply_quantize(buf, buffer, N);
    p->run(buf, buffer);
    p->sync();
}

void *getBuffer(size_t N)
{
    void *tmp;
    checked(hipHostMalloc(&tmp, N));
    return tmp;
}

void freeBuffer(void *b)
{
    if(b)
        checked(hipHostFree(b));
}
